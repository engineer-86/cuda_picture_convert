#include "hip/hip_runtime.h"
//
// Created by Konrad Münch on 08.12.2023.
//

#include <algorithm>
#include "HostSystem.cuh"
#include "../helper/HelperFunctions.cuh"
#include <cmath>
#include <omp.h>
#include <iostream>


__host__ void ConvertRGBtoHSV(uchar3 *input, float3 *output, int width, int height) {


#pragma omp parallel for
    for (int y = 0; y < height; ++y) {

        for (int x = 0; x < width; ++x) {
            unsigned int idx = y * width + x;
            uchar3 rgb = input[idx];


            float B = static_cast<float>(rgb.z) / 255.0f;
            float G = static_cast<float>(rgb.y) / 255.0f;
            float R = static_cast<float>(rgb.x) / 255.0f;

            float c_max = std::max({R, G, B});
            float c_min = std::min({R, G, B});
            float diff = c_max - c_min;

            float H = 0;
            if (diff != 0) {
                if (c_max == R) {
                    H = fmodf((60 * ((G - B) / diff) + 360), 360);
                } else if (c_max == G) {
                    H = fmodf((60 * ((B - R) / diff) + 120), 360);
                } else if (c_max == B) {
                    H = fmodf((60 * ((R - G) / diff) + 240), 360);
                }
            }

            float S, V;
            if (c_max == 0) {
                S = 0;
            } else {
                S = diff / c_max * 255;
            }
            V = c_max * 255;

            output[idx] = make_float3(H, S, V);
        }
    }
}

__host__ void AddBoxBlur(uchar3 *input, uchar3 *output, int width, int height) {
    int blurRadius = 10;


#pragma omp parallel for
    for (int y = 0; y < height; y++) {

        for (int x = 0; x < width; x++) {
            unsigned int idx = y * width + x;

            float3 sum = {0, 0, 0};
            int count = 0;

            for (int i = -blurRadius; i <= blurRadius; i++) {
                for (int j = -blurRadius; j <= blurRadius; j++) {
                    int x1 = x + i;
                    int y1 = y + j;

                    if (x1 >= 0 && x1 < width && y1 >= 0 && y1 < height) {
                        unsigned int idx1 = y1 * width + x1;
                        uchar3 inPixel = input[idx1];
                        sum.x += inPixel.x;
                        sum.y += inPixel.y;
                        sum.z += inPixel.z;
                        count++;
                    }
                }
            }

            output[idx] = make_uchar3(sum.z / count, sum.y / count, sum.x / count);
        }
    }
}
