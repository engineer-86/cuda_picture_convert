#include "hip/hip_runtime.h"
//
// Created by Konrad Münch on 08.12.2023.
//

#include <algorithm>
#include "HostSystem.cuh"
#include <cmath>
#include <omp.h>
#include <iostream>


__host__ void ConvertRGBtoHSV(uchar3 *input, float3 *output, int width, int height) {
    omp_set_dynamic(0);
    omp_set_num_threads(20);

#pragma omp parallel for
    for (int y = 0; y < height; ++y) {

        int threadID = omp_get_thread_num();
#pragma omp critical
        {
            static int numThreadsReported = 0;
            if (numThreadsReported++ < omp_get_num_threads()) {
                printf("Thread-ID: %d from %d Threads \n", threadID, omp_get_num_threads());
            }
        }
        for (int x = 0; x < width; ++x) {
            unsigned int idx = y * width + x;
            uchar3 rgb = input[idx];


            float B = static_cast<float>(rgb.z) / 255.0f;
            float G = static_cast<float>(rgb.y) / 255.0f;
            float R = static_cast<float>(rgb.x) / 255.0f;

            float c_max = std::max({R, G, B});
            float c_min = std::min({R, G, B});
            float diff = c_max - c_min;

            float H = 0;
            if (diff != 0) {
                if (c_max == R) {
                    H = fmodf((60 * ((G - B) / diff) + 360), 360);
                } else if (c_max == G) {
                    H = fmodf((60 * ((B - R) / diff) + 120), 360);
                } else if (c_max == B) {
                    H = fmodf((60 * ((R - G) / diff) + 240), 360);
                }
            }

            float S, V;
            if (c_max == 0) {
                S = 0;
            } else {
                S = diff / c_max * 255;
            }
            V = c_max * 255;

            output[idx] = make_float3(H, S, V);
        }
    }
}

__host__ void AddBoxBlur(uchar3 *input, uchar3 *output, int width, int height) {
    int blurRadius = 10;
    omp_set_dynamic(0);
    omp_set_num_threads(20);

#pragma omp parallel for
    for (int y = 0; y < height; y++) {

        int threadID = omp_get_thread_num();
#pragma omp critical
        {
            static int numThreadsReported = 0;
            if (numThreadsReported++ < omp_get_num_threads()) {
                printf("Thread-ID: %d from %d Threads \n", threadID, omp_get_num_threads());
            }
        }
        for (int x = 0; x < width; x++) {
            unsigned int idx = y * width + x;

            float3 sum = {0, 0, 0};
            int count = 0;

            for (int i = -blurRadius; i <= blurRadius; i++) {
                for (int j = -blurRadius; j <= blurRadius; j++) {
                    int x1 = x + i;
                    int y1 = y + j;

                    if (x1 >= 0 && x1 < width && y1 >= 0 && y1 < height) {
                        unsigned int idx1 = y1 * width + x1;
                        uchar3 inPixel = input[idx1];
                        sum.x += inPixel.x;
                        sum.y += inPixel.y;
                        sum.z += inPixel.z;
                        count++;
                    }
                }
            }

            output[idx] = make_uchar3(sum.z / count, sum.y / count, sum.x / count);
        }
    }
}
