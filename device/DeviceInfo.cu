//
// Created by KonradMünch on 16.10.2023.
//
#include <iostream>
#include "DeviceInfo.cuh"

void getDeviceInfo() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp{};
        hipGetDeviceProperties(&deviceProp, device);
        std::cout << "Device " << device << ": " << deviceProp.name << std::endl;
        std::cout << "  CUDA Version: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  V-RAM: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Multiprocessor Count: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Max Threads Per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "  Clock rate:" << deviceProp.clockRate / 1000 << "Mhz" << std::endl;
    }
}

