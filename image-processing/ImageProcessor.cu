#include "hip/hip_runtime.h"
//
// Created by Konrad Münch on 04.11.2023.
//

#include "ImageProcessor.cuh"
#include <iostream>
#include "../device/DeviceKernel.cuh"
#include <chrono>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/types_c.h>
#include "../host/HostSystem.cuh"
#include "../helper/GpuPowerMonitor.cuh"
#include "../helper/HelperFunctions.cuh"
#include "../helper/GpuPowerMonitorThread.cuh"


ImageProcessor::ImageProcessor() : total_time_blur_(0.0),
                                   total_time_hsv_(0.0),
                                   mode_("CPU"),
                                   output_kind_("RGB") {

}

ImageProcessor::~ImageProcessor() {}

void ImageProcessor::LoadImage(const std::string &file_name) {
    this->image_rgb_ = cv::imread(file_name, cv::IMREAD_UNCHANGED);
    if (image_rgb_.empty()) {
        throw std::runtime_error("Error: Unable to read the image file.");
    }
    rows_ = image_rgb_.rows;
    columns_ = image_rgb_.cols;
}


void ImageProcessor::SaveImage(const std::string &file_path, const std::string &output_kind, bool mode) {
    std::string calc_mode = mode ? "cuda" : "cpu";

    if (output_kind == "blur") {
        cv::imwrite(file_path + calc_mode + "_" + "blur.jpeg", this->image_blur_);
    } else if
            (output_kind == "hsv") {
        cv::imwrite(file_path + calc_mode + "_"  "hsv.jpeg", this->image_hsv_);
    } else {
        cv::imwrite(file_path + calc_mode + "_"  "rgb.jpeg", this->image_rgb_);
    }
}

const cv::Mat &ImageProcessor::GetImage() const {
    return this->image_rgb_;
}

void ImageProcessor::ProcessImageCUDA(const std::string &input_picture_path,
                                      const std::string &output_picture_path,
                                      bool is_gpu_available, int runId,
                                      std::vector<ProcessingInfo> &infos) {


    std::cout << "Try to load Image: " << input_picture_path << std::endl;
    this->LoadImage(input_picture_path);

    std::cout << "Convert Pictures to HSV and BLUR with GPU." << std::endl;
    infos.push_back(this->ConvertRGBtoHSVCuda(runId));
    infos.push_back(this->AddBoxBlurCuda(runId));


    std::cout << "Save Image: " << output_picture_path << std::endl;
    this->SaveImage(output_picture_path, "hsv", is_gpu_available);
    this->SaveImage(output_picture_path, "blur", is_gpu_available);;
}

void ImageProcessor::ProcessImageCPU(const std::string &input_picture_path,
                                     const std::string &output_picture_path,
                                     bool is_gpu_available, int runId,
                                     std::vector<ProcessingInfo> &infos) {

    std::cout << "Try to load Image: " << input_picture_path << std::endl;
    this->LoadImage(input_picture_path);

    std::cout << "Convert Pictures to HSV and BLUR with CPU." << std::endl;
    infos.push_back(this->ConvertRGBtoHSVHost(runId));
    infos.push_back(this->AddBoxBlurHost(runId));


    std::cout << "Save Image: " << output_picture_path << std::endl;
    this->SaveImage(output_picture_path, "hsv", is_gpu_available);
    this->SaveImage(output_picture_path, "blur", is_gpu_available);
}

ProcessingInfo ImageProcessor::ConvertRGBtoHSVCuda(int runId) {

    cv::Mat image = this->GetImage();
    int width = this->columns_;
    int height = this->rows_;
    uchar3 *d_input;
    float3 *d_output;


    hipMalloc(&d_input, width * height * sizeof(uchar3));
    hipMalloc(&d_output, width * height * sizeof(float3));
    hipMemcpy(d_input, image.ptr<uchar>(0), width * height * sizeof(uchar3),
               hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);


    hipEvent_t start, stop;
    float duration;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );

    auto start_time = std::chrono::high_resolution_clock::now();
    GpuPowerMonitorThread gpuPowerMonitorThread;
    gpuPowerMonitorThread.startMonitoring();

    // 8.1.2. Using CUDA GPU Timers
    // https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html

    ConvertRGBtoHSVKernel<<<gridSize, blockSize>>>(d_input, d_output, width, height);

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime(&duration, start, stop );
    hipEventDestroy( start );
    hipEventDestroy( stop );

    auto finish = std::chrono::high_resolution_clock::now();

    this->AddTimeToCUDAHSV(duration);
    this->SetTotalTimeHSV(duration);

    hipGetLastError();
    hipDeviceSynchronize();
    // stop power measure thread
    gpuPowerMonitorThread.stopMonitoring();


    auto timestamp = HelperFunctions::getCurrentTimestamp();

    cv::Mat hsv_image(height, width, CV_32FC3);
    hipMemcpy(hsv_image.ptr<float>(0), d_output, width * height * sizeof(float3),
               hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    this->image_hsv_ = hsv_image;
    auto powers = gpuPowerMonitorThread.getPowerReadings();

    return ProcessingInfo(runId, timestamp, "hsv", "CUDA", duration,
                          gpuPowerMonitorThread.getPowerReadings());
}

#include <chrono>


ProcessingInfo ImageProcessor::AddBoxBlurCuda(int runId) {

    cv::Mat image = this->GetImage();
    int width = this->columns_;
    int height = this->rows_;
    uchar3 *d_input;
    uchar3 *d_output;

    hipMalloc(&d_input, width * height * sizeof(uchar3));
    hipMalloc(&d_output, width * height * sizeof(uchar3));
    hipMemcpy(d_input, image.ptr<uchar>(0), width * height * sizeof(uchar3),
               hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);


    auto start = std::chrono::high_resolution_clock::now();
    GpuPowerMonitor gpuPowerMonitor;
    float startPower = gpuPowerMonitor.getPowerUsage(0);

    AddBoxBlurKernel<<<gridSize, blockSize>>>(d_input, d_output, width, height);


    auto finish = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration<double>(finish - start).count();
    this->AddTimeToCUDABlur(duration);
    this->SetTotalTimeBlur(duration);

    hipGetLastError();
    hipDeviceSynchronize();
    float endPower = gpuPowerMonitor.getPowerUsage(0); // GPU-Index 0
    auto timestamp = HelperFunctions::getCurrentTimestamp();
    float averagePower = (startPower + endPower) / 2.0f;


    cv::Mat blurred_image(height, width, CV_8UC3);
    hipMemcpy(blurred_image.ptr<float>(0), d_output, width * height * sizeof(uchar3),
               hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    this->image_blur_ = blurred_image;


    return ProcessingInfo(runId, timestamp, "blur", "CUDA", duration, averagePower);

}


ProcessingInfo ImageProcessor::ConvertRGBtoHSVHost(int runId) {
    cv::Mat image = this->GetImage();
    int width = image.cols;
    int height = image.rows;

    std::vector<uchar3> input = this->ImageToVector();
    std::vector<float3> output(width * height);

    auto start = std::chrono::high_resolution_clock::now();

    ConvertRGBtoHSV(input.data(), output.data(), width, height);
    auto timestamp = HelperFunctions::getCurrentTimestamp();
    auto finish = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration<double>(finish - start).count();

    this->AddTimeToCPUHSV(duration);
    this->SetTotalTimeHSV(duration);


    this->VectorToImage(output, "hsv");
    return ProcessingInfo(runId, timestamp, "hsv", "CPU", duration, 0.0);
}

ProcessingInfo ImageProcessor::AddBoxBlurHost(int runId) {
    cv::Mat image = this->GetImage();
    int width = image.cols;
    int height = image.rows;

    std::vector<uchar3> input = this->ImageToVector();
    std::vector<uchar3> output(width * height);

    auto start = std::chrono::high_resolution_clock::now();

    AddBoxBlur(input.data(), output.data(), width, height);
    auto timestamp = HelperFunctions::getCurrentTimestamp();
    auto finish = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration<double>(finish - start).count();

    this->AddTimeToCPUBlur(duration);
    this->SetTotalTimeBlur(duration);
    this->VectorToImage(output, "blur");
    return ProcessingInfo(runId, timestamp, "blur", "CPU", duration, 0.0);
}


std::vector<uchar3> ImageProcessor::ImageToVector() {
    cv::Mat image = this->GetImage();
    int width = image.cols;
    int height = image.rows;

    // input and output vectors
    std::vector<uchar3> input(width * height);

    // convert mat-pic to cv vector with 2 nested for loops

    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; x++) {
            cv::Vec3b pixel = image.at<cv::Vec3b>(y, x);
            input[y * width + x] = make_uchar3(pixel[2], pixel[1], pixel[0]); // BGR to RGB
        }
    }

    return input;
}

template<typename T>
void ImageProcessor::VectorToImage(const std::vector<T> &output, const std::string &convert_mode) {
    cv::Mat image = this->GetImage();
    int width = image.cols;
    int height = image.rows;


    cv::Mat *target_image = nullptr;
    if (convert_mode == "hsv") {
        target_image = &this->image_hsv_;
    } else if (convert_mode == "blur") {
        target_image = &this->image_blur_;
    }

    if (!target_image) {

        throw std::invalid_argument("Please provide convert mode: hsv or blur");
    }


    if (std::is_same<T, uchar3>::value) {
        *target_image = cv::Mat(height, width, CV_8UC3);
    } else if (std::is_same<T, float3>::value) {
        *target_image = cv::Mat(height, width, CV_32FC3);
    }


    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            T hsv = output[y * width + x];
            if (std::is_same<T, uchar3>::value) {
                target_image->at<cv::Vec3b>(y, x) = cv::Vec3b(hsv.x, hsv.y, hsv.z);
            } else if (std::is_same<T, float3>::value) {
                target_image->at<cv::Vec3f>(y, x) = cv::Vec3f(hsv.x, hsv.y, hsv.z);
            }
        }
    }
}

double ImageProcessor::getTotalTimeBlur() const {
    return total_time_blur_;
}

double ImageProcessor::getTotalTimeHSV() const {
    return total_time_hsv_;
}

void ImageProcessor::SetTotalTimeHSV(double total_time) {
    this->total_time_hsv_ += total_time;
}

void ImageProcessor::SetTotalTimeBlur(double total_time) {
    this->total_time_blur_ += total_time;
}

void ImageProcessor::AddTimeToCUDABlur(double time) {
    single_times_cuda_blur_.push_back(time);
}

void ImageProcessor::AddTimeToCUDAHSV(double time) {
    single_times_cuda_hsv_.push_back(time);
}

std::vector<double> ImageProcessor::GetTimesCUDABlur() const {
    return single_times_cuda_blur_;
}

std::vector<double> ImageProcessor::GetTimesCUDAHSV() const {
    return single_times_cuda_hsv_;
}

void ImageProcessor::AddTimeToCPUBlur(double time) {
    single_times_cuda_blur_.push_back(time);
}

void ImageProcessor::AddTimeToCPUHSV(double time) {
    single_times_cuda_hsv_.push_back(time);
}

std::vector<double> ImageProcessor::GetTimesCPUBlur() const {
    return single_times_cuda_blur_;
}

std::vector<double> ImageProcessor::GetTimesCPUHSV() const {
    return single_times_cuda_hsv_;
}

const std::string &ImageProcessor::getMode() const {
    return mode_;
}

void ImageProcessor::setMode(const std::string &mode) {
    mode_ = mode;
}

const std::string &ImageProcessor::getOutputKind() const {
    return output_kind_;
}

void ImageProcessor::setOutputKind(const std::string &outputKind) {
    output_kind_ = outputKind;
}

